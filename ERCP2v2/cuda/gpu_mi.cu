#include "hip/hip_runtime.h"
/*
 Copyright Ramtin Shams (hereafter referred to as 'the author'). All rights 
 reserved. **Citation required in derived works or publications** 
 
 NOTICE TO USER:   
 
 Users and possessors of this source code are hereby granted a nonexclusive, 
 royalty-free license to use this source code for non-commercial purposes only, 
 as long as the author is appropriately acknowledged by inclusion of this 
 notice in derived works and citation of appropriate publication(s) listed 
 at the end of this notice in any derived works or publications that use 
 or have benefited from this source code in its entirety or in part.
   
 
 THE AUTHOR MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 IMPLIED WARRANTY OF ANY KIND.  THE AUTHOR DISCLAIMS ALL WARRANTIES WITH 
 REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 OR PERFORMANCE OF THIS SOURCE CODE.  
 
 Relevant publication(s):
	@inproceedings{Shams_ICSPCS_2007,
		author        = "R. Shams and R. A. Kennedy",
		title         = "Efficient Histogram Algorithms for {NVIDIA} {CUDA} Compatible Devices",
		booktitle     = "Proc. Int. Conf. on Signal Processing and Communications Systems ({ICSPCS})",
		address       = "Gold Coast, Australia",
		month         = dec,
		year          = "2007",
		pages         = "418-422",
	}

	@inproceedings{Shams_DICTA_2007a,
		author        = "R. Shams and N. Barnes",
		title         = "Speeding up Mutual Information Computation Using {NVIDIA} {CUDA} Hardware",
		booktitle     = "Proc. Digital Image Computing: Techniques and Applications ({DICTA})",
		address       = "Adelaide, Australia",
		month         = dec,
		year          = "2007",
		pages         = "555-560",
		doi           = "10.1109/DICTA.2007.4426846",
	};
*/
#ifndef _GPU_MI_H_
#define _GPU_MI_H_

#include "gpu_basics.h"

__device__ float inlineEntropyUnary(float a)
{
	if (a > 0.0f)
		return -a * logf(a);
	else
		return 0.0f;
}

GPU_UNARY(EntropyUnary)

#endif