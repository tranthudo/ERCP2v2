#include "hip/hip_runtime.h"
/*
 Copyright Ramtin Shams (hereafter referred to as 'the author'). All rights 
 reserved. **Citation required in derived works or publications** 
 
 NOTICE TO USER:   
 
 Users and possessors of this source code are hereby granted a nonexclusive, 
 royalty-free license to use this source code for non-commercial purposes only, 
 as long as the author is appropriately acknowledged by inclusion of this 
 notice in derived works and citation of appropriate publication(s) listed 
 at the end of this notice in any derived works or publications that use 
 or have benefited from this source code in its entirety or in part.
   
 
 THE AUTHOR MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 IMPLIED WARRANTY OF ANY KIND.  THE AUTHOR DISCLAIMS ALL WARRANTIES WITH 
 REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 OR PERFORMANCE OF THIS SOURCE CODE.  
 
 Relevant publication(s):
	@inproceedings{Shams_ICSPCS_2007,
		author        = "R. Shams and R. A. Kennedy",
		title         = "Efficient Histogram Algorithms for {NVIDIA} {CUDA} Compatible Devices",
		booktitle     = "Proc. Int. Conf. on Signal Processing and Communications Systems ({ICSPCS})",
		address       = "Gold Coast, Australia",
		month         = dec,
		year          = "2007",
		pages         = "418-422",
	}

	@inproceedings{Shams_DICTA_2007a,
		author        = "R. Shams and N. Barnes",
		title         = "Speeding up Mutual Information Computation Using {NVIDIA} {CUDA} Hardware",
		booktitle     = "Proc. Digital Image Computing: Techniques and Applications ({DICTA})",
		address       = "Adelaide, Australia",
		month         = dec,
		year          = "2007",
		pages         = "555-560",
		doi           = "10.1109/DICTA.2007.4426846",
	};
*/

// includes, system
#include <stdlib.h>
#include <tchar.h>
#include <stdarg.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>
#include <hip/driver_types.h>
#include "cuda_basics.h"
#include "cuda_hist.h"
#include "cuda_mi.h"

// includes, kernels
#include "gpu_mi.cu"

extern "C" float cudaMIa(float *src1, float *src2, int length, int xbins, int ybins, double &time, cudaHistOptions *p_options /*= NULL*/, bool device /*= false*/)
{
	dim3 grid, block;
	int size = length * sizeof(float);
	//Device memory pointers
	float *d_src1, *d_src2, *d_hist, *d_rows, *d_cols;
	time = 0;
	unsigned int hTimer;
    CUT_SAFE_CALL(cutCreateTimer(&hTimer));
	TIMER_CREATE;

	TIMER_START;
	if (!device)
	{
		//Allocate data on the device
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src1, size));
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src2, size));

		//Copy src data to device memory
		CUDA_SAFE_CALL(hipMemcpy(d_src1, src1, size, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_src2, src2, size, hipMemcpyHostToDevice));
	}
	else
	{
		d_src1 = src1; d_src2 = src2;
	}

    CUDA_SAFE_CALL(hipDeviceSynchronize());								
    CUT_SAFE_CALL(cutStartTimer(hTimer));								

	CUDA_SAFE_CALL(hipMalloc((void**) &d_hist, xbins * ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_rows, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_rows, 0, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_cols, xbins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_cols, 0, xbins * sizeof(float)));
	TIMER_PRINT("Loading data", 0);

	cudaHist2Da(d_src1, d_src2, d_hist, length, xbins, ybins, p_options, true);					//No need to initialize d_hist, it will be done by cudaHist

	int totalSamples = length;
	//Caculate MI from the joint histogram
	// H(X,Y)
	float Hxy = cudaEntropy(d_hist, xbins * ybins, true) / totalSamples + logf((float)totalSamples);

	// H(X)
	cudaSumAlongCols(d_hist, d_cols, xbins, ybins, true);
	float Hx = cudaEntropy(d_cols, xbins, true) / totalSamples + logf((float)totalSamples);

	// H(Y)
	cudaSumAlongRows(d_hist, d_rows, xbins, ybins, true);
	float Hy = cudaEntropy(d_rows, ybins, true) / totalSamples + logf((float)totalSamples);

	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(cutStopTimer(hTimer));
	time = cutGetTimerValue(hTimer);
	CUT_SAFE_CALL(cutDeleteTimer(hTimer));

	if (!device)
	{
		CUDA_SAFE_CALL(hipFree(d_src1));
		CUDA_SAFE_CALL(hipFree(d_src2));
	}

	CUDA_SAFE_CALL(hipFree(d_hist));
	CUDA_SAFE_CALL(hipFree(d_rows));
	CUDA_SAFE_CALL(hipFree(d_cols));
	TIMER_DELETE;

	return Hx + Hy - Hxy;
}

extern "C" float cudaMIb(float *src1, float *src2, int length, int xbins, int ybins, double &time, cudaHistOptions *p_options /*= NULL*/, bool device /*= false*/)
{
	dim3 grid, block;
	int size = length * sizeof(float);
	//Device memory pointers
	float *d_src1, *d_src2, *d_hist, *d_rows, *d_cols;
	time = 0;
	unsigned int hTimer;
    CUT_SAFE_CALL(cutCreateTimer(&hTimer));
	TIMER_CREATE;

	TIMER_START;
	if (!device)
	{
		//Allocate data on the device
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src1, size));
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src2, size));

		//Copy src data to device memory
		CUDA_SAFE_CALL(hipMemcpy(d_src1, src1, size, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_src2, src2, size, hipMemcpyHostToDevice));
	}
	else
	{
		d_src1 = src1; d_src2 = src2;
	}

    CUDA_SAFE_CALL(hipDeviceSynchronize());								
    CUT_SAFE_CALL(cutStartTimer(hTimer));								

	CUDA_SAFE_CALL(hipMalloc((void**) &d_hist, xbins * ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_rows, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_rows, 0, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_cols, xbins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_cols, 0, xbins * sizeof(float)));
	TIMER_PRINT("Loading data", 0);

	cudaHist2Db(d_src1, d_src2, d_hist, length, xbins, ybins, p_options, true);					//No need to initialize d_hist, it will be done by cudaHist

	int totalSamples = length;
	//Caculate MI from the joint histogram
	// H(X,Y)
	float Hxy = cudaEntropy(d_hist, xbins * ybins, true) / totalSamples + logf((float)totalSamples);

	// H(X)
	cudaSumAlongCols(d_hist, d_cols, xbins, ybins, true);
	float Hx = cudaEntropy(d_cols, xbins, true) / totalSamples + logf((float)totalSamples);

	// H(Y)
	cudaSumAlongRows(d_hist, d_rows, xbins, ybins, true);
	float Hy = cudaEntropy(d_rows, ybins, true) / totalSamples + logf((float)totalSamples);

	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(cutStopTimer(hTimer));
	time = cutGetTimerValue(hTimer);
	CUT_SAFE_CALL(cutDeleteTimer(hTimer));

	if (!device)
	{
		CUDA_SAFE_CALL(hipFree(d_src1));
		CUDA_SAFE_CALL(hipFree(d_src2));
	}

	CUDA_SAFE_CALL(hipFree(d_hist));
	CUDA_SAFE_CALL(hipFree(d_rows));
	CUDA_SAFE_CALL(hipFree(d_cols));
	TIMER_DELETE;

	return Hx + Hy - Hxy;
}

extern "C" float cudaMI_Approx(float *src1, float *src2, int length, int xbins, int ybins, double &time, cudaHistOptions *p_options /*= NULL*/, bool device /*= false*/)
{
	dim3 grid, block;
	int size = length * sizeof(float);
	//Device memory pointers
	float *d_src1, *d_src2, *d_hist, *d_rows, *d_cols;
	time = 0;
	unsigned int hTimer;
    CUT_SAFE_CALL(cutCreateTimer(&hTimer));
	TIMER_CREATE;

	TIMER_START;
	if (!device)
	{
		//Allocate data on the device
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src1, size));
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src2, size));

		//Copy src data to device memory
		CUDA_SAFE_CALL(hipMemcpy(d_src1, src1, size, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_src2, src2, size, hipMemcpyHostToDevice));
	}
	else
	{
		d_src1 = src1; d_src2 = src2;
	}

    CUDA_SAFE_CALL(hipDeviceSynchronize());								
    CUT_SAFE_CALL(cutStartTimer(hTimer));								

	CUDA_SAFE_CALL(hipMalloc((void**) &d_hist, xbins * ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_rows, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_rows, 0, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_cols, xbins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_cols, 0, xbins * sizeof(float)));
	TIMER_PRINT("Loading data", 0);

	cudaHist2D_Approx(d_src1, d_src2, d_hist, length, xbins, ybins, p_options, true);					//No need to initialize d_hist, it will be done by cudaHist

	int totalSamples = (int) cudaSum(d_hist, xbins * ybins, true);
	//Caculate MI from the joint histogram
	// H(X,Y)
	float Hxy = cudaEntropy(d_hist, xbins * ybins, true) / totalSamples + logf((float)totalSamples);

	// H(X)
	cudaSumAlongCols(d_hist, d_cols, xbins, ybins, true);
	float Hx = cudaEntropy(d_cols, xbins, true) / totalSamples + logf((float)totalSamples);

	// H(Y)
	cudaSumAlongRows(d_hist, d_rows, xbins, ybins, true);
	float Hy = cudaEntropy(d_rows, ybins, true) / totalSamples + logf((float)totalSamples);

	if (!device)
	{
		CUDA_SAFE_CALL(hipFree(d_src1));
		CUDA_SAFE_CALL(hipFree(d_src2));
	}

	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(cutStopTimer(hTimer));
	time = cutGetTimerValue(hTimer);
	CUT_SAFE_CALL(cutDeleteTimer(hTimer));

	CUDA_SAFE_CALL(hipFree(d_hist));
	CUDA_SAFE_CALL(hipFree(d_rows));
	CUDA_SAFE_CALL(hipFree(d_cols));
	TIMER_DELETE;

	return Hx + Hy - Hxy;
}

extern "C" float cudaEntropy(float *src, int length, bool device /*= false*/)
{
	float *d_src, *d_dst;
	int size = length * sizeof(float);
	if (!device)
	{
		CUDA_SAFE_CALL(hipMalloc((void**)&d_src, size));
		//Copy src data to device memory
		CUDA_SAFE_CALL(hipMemcpy(d_src, src, size, hipMemcpyHostToDevice));
	}
	else
	{
		d_src = src;
	}
	CUDA_SAFE_CALL(hipMalloc((void**)&d_dst, size));

	cudaEntropyUnary(d_src, d_dst, length, true);
	float res = cudaSum(d_dst, length, true);

	if (!device)																				
		CUDA_SAFE_CALL(hipFree(d_src));
	CUDA_SAFE_CALL(hipFree(d_dst));

	return res;
}

CUDA_UNARY(EntropyUnary);
